#include "hip/hip_runtime.h"
#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif // _WIN32
    exit(EXIT_FAILURE);
#endif // ERRORCHECK
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
static BVHNode* dev_bvh = NULL;

void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    hipMalloc(&dev_bvh, scene->bvh.size() * sizeof(BVHNode));
    hipMemcpy(dev_bvh, scene->bvh.data(), scene->bvh.size() * sizeof(BVHNode), hipMemcpyHostToDevice);

    checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);
    hipFree(dev_bvh);
    checkCUDAError("pathtraceFree");
}

__device__ glm::vec2 SampleUniformDisk(float u, float v) {
    float dx = sqrt(u) * cos(TWO_PI * v);
    float dy = sqrt(u) * sin(TWO_PI * v);
    return glm::vec2(dx, dy) * 2.0f - glm::vec2(1., 1.);
}
/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];

        segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);
        
        thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, traceDepth);
        thrust::uniform_real_distribution<float> u01(0.0f, 1.0f);

        float randX = 0.0f, randY = 0.0f;

        #ifdef ANTI_ALIAS
        randX = u01(rng) - 0.5f;
        randY = u01(rng) - 0.5f;
        #endif

        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f + randX)
            - cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f + randY)
        );
        
        #ifdef DOF
        glm::vec2 rand_dir = cam.aperture * SampleUniformDisk(u01(rng), u01(rng));
        segment.ray.origin = cam.position + cam.right * rand_dir.x + cam.up * rand_dir.y;

        glm::vec3 focus_point = cam.position + segment.ray.direction * cam.focus_dis;
        segment.ray.direction = glm::normalize(focus_point - segment.ray.origin);
        #endif

        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}

__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    Geom* geoms,
    BVHNode* bvh,
    int geoms_size,
    ShadeableIntersection* intersections)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment pathSegment = pathSegments[path_index];

        float t;
        glm::vec3 intersect_point;
        glm::vec3 normal;
        float t_min = FLT_MAX;
        int hit_geom_index = -1;
        bool outside = true;

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;

        #ifdef BVH_NUM
        int stack[20];
        stack[0] = 0;
        int top = 0;

        while(top >= 0){
            int cur_node = stack[top--];

            if(BVHIntersectionTest(bvh[cur_node], pathSegment.ray)) {
                if(bvh[cur_node].isLeaf){
                    for (int i = bvh[cur_node].geo_st; i < bvh[cur_node].geo_ed; i++) {
                        Geom& geom = geoms[i];

                        if (geom.type == CUBE)
                        {
                            t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                        }
                        else if (geom.type == SPHERE)
                        {
                            t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                        }
                        else if(geom.type == TRIANGLE){
                            t = triIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                        }
                        if (t > 0.0f && t_min > t)
                        {
                            t_min = t;
                            hit_geom_index = i;
                            intersect_point = tmp_intersect;
                            normal = tmp_normal;
                        }
                    }
                }
                else{
                    stack[++top] = bvh[cur_node].rightNode;
                    stack[++top] = bvh[cur_node].leftNode;
                }
            }
        }

        #else

        for (int i = 0; i < geoms_size; i++)
        {
            Geom& geom = geoms[i];

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            else if(geom.type == TRIANGLE){
                t = triIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                hit_geom_index = i;
                intersect_point = tmp_intersect;
                normal = tmp_normal;
            }
        }
        #endif

        if (hit_geom_index == -1)
        {
            intersections[path_index].t = -1.0f;
        }
        else
        {
            // The ray hits something
            intersections[path_index].t = t_min;
            intersections[path_index].materialId = geoms[hit_geom_index].materialid;
            intersections[path_index].surfaceNormal = normal;
        }

    }
}

__global__ void shadeMaterial(
    int iter,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(pathSegments[idx].remainingBounces <= 0) return;
    if (idx < num_paths)
    {
        ShadeableIntersection intersection = shadeableIntersections[idx];
        if (intersection.t > 0.0f) // if the intersection exists...
        {
          // Set up the RNG
          // LOOK: this is how you use thrust's RNG! Please look at
          // makeSeededRandomEngine as well.
            thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
            thrust::uniform_real_distribution<float> u01(0, 1);

            Material material = materials[intersection.materialId];
            glm::vec3 materialColor = material.color;

            // If the material indicates that the object was a light, "light" the ray
            if (material.emittance > 0.0f) {
                pathSegments[idx].color *= (materialColor * material.emittance) * glm::dot(glm::normalize(-pathSegments[idx].ray.direction), glm::normalize(intersection.surfaceNormal));
                pathSegments[idx].remainingBounces = 0;
            }
            else {
                glm::vec3 inter_pos = getPointOnRay(pathSegments[idx].ray, intersection.t);
                scatterRay(pathSegments[idx], inter_pos, intersection.surfaceNormal, material, rng);

            #ifdef ROULETTE
                float russian = u01(rng);
                if(russian > ROULETTE){
                    pathSegments[idx].remainingBounces = 0;
                }
                else{
                    pathSegments[idx].color /= ROULETTE;
                    pathSegments[idx].remainingBounces--;
                }
            #else
                pathSegments[idx].remainingBounces--;
            #endif


                if(pathSegments[idx].remainingBounces == 0){
                    pathSegments[idx].color = glm::vec3(0.0f, 0.0f, 0.0f);
                }
            }
        }
        else {
            pathSegments[idx].color = glm::vec3(0.0f, 0.0f, 0.0f);
            pathSegments[idx].remainingBounces = 0;
        }
    }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += iterationPath.color;
    }
}

struct deadLight
{
    __host__ __device__
    bool operator()(const PathSegment &p) const
    {
        return (p.remainingBounces > 0);
    }
};

struct sortMaterial
{
    __host__ __device__
    bool operator()(const ShadeableIntersection &a, const ShadeableIntersection &b) const {
        return a.materialId < b.materialId;
    }
};



/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // perform one iteration of path tracing

    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");
    hipDeviceSynchronize();

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths;

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    bool iterationComplete = false;
    while (!iterationComplete)
    {
        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>> (
            depth,
            num_paths,
            dev_paths,
            dev_geoms,
            dev_bvh,
            hst_scene->geoms.size(),
            dev_intersections
        );
        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();
        depth++;

        shadeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(
            iter,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials
        );
        checkCUDAError("shading");
        hipDeviceSynchronize();

        PathSegment *new_end = thrust::partition(
            thrust::device,
            dev_paths,
            dev_paths + num_paths,
            deadLight());

        num_paths = new_end - dev_paths;
        iterationComplete = (num_paths == 0);

        #ifdef SORT_MATERIAL
        thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + num_paths, dev_paths, sortMaterial());
        #endif

        if (guiData != NULL)
        {
            guiData->TracedDepth = depth;
        }
    }

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather<<<numBlocksPixels, blockSize1d>>>(pixelcount, dev_image, dev_paths); // trap here, use pixelcount instead of num_path

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
